#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <ctime>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

#include <>
#include <hip/device_functions.h>

// Custom header 
#include "kernel.h"

//#define INFO 1
#ifdef  INFO
__device__ u32 totalThreadCount = 0;
__device__ ull totalEncryptions = 0;
__device__ u32 maxThreadIndex = 0;
#endif // INFO

// Basic exhaustive search
// 4 Tables
__global__ void exhaustiveSearch(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t1G, u32* t2G, u32* t3G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE];
	__shared__ u32 t1S[TABLE_SIZE];
	__shared__ u32 t2S[TABLE_SIZE];
	__shared__ u32 t3S[TABLE_SIZE];
	__shared__ u32 t4S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t0S[threadIdx.x] = t0G[threadIdx.x];
		t1S[threadIdx.x] = t1G[threadIdx.x];
		t2S[threadIdx.x] = t2G[threadIdx.x];
		t3S[threadIdx.x] = t3G[threadIdx.x];
		t4S[threadIdx.x] = t4G[threadIdx.x];

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}
		 
		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	ull threadRangeStart = (ull)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		//if (threadIndex == 0 && rangeCount == 0) {
		//	printf("--Round: %d\n", 0);
		//	printf("%08x%08x%08x%08x\n", s0, s1, s2, s3);
		//	printf("-- Round Key\n");
		//	printf("%08x%08x%08x%08x\n", rk0, rk1, rk2, rk3);
		//}

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			// TODO: temp & 0xff000000
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
				(t4S[(temp >>  8) & 0xff] & 0x00ff0000) ^
				(t4S[(temp      ) & 0xff] & 0x0000ff00) ^
				(t4S[(temp >> 24)       ] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24] ^ t1S[(s1 >> 16) & 0xFF] ^ t2S[(s2 >> 8) & 0xFF] ^ t3S[s3 & 0xFF] ^ rk0;
			t1 = t0S[s1 >> 24] ^ t1S[(s2 >> 16) & 0xFF] ^ t2S[(s3 >> 8) & 0xFF] ^ t3S[s0 & 0xFF] ^ rk1;
			t2 = t0S[s2 >> 24] ^ t1S[(s3 >> 16) & 0xFF] ^ t2S[(s0 >> 8) & 0xFF] ^ t3S[s1 & 0xFF] ^ rk2;
			t3 = t0S[s3 >> 24] ^ t1S[(s0 >> 16) & 0xFF] ^ t2S[(s1 >> 8) & 0xFF] ^ t3S[s2 & 0xFF] ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

			//if (threadIndex == 0 && rangeCount == 0) {
			//	printf("--Round: %d\n", roundCount);
			//	printf("%08x%08x%08x%08x\n", s0, s1, s2, s3);
			//	printf("-- Round Key\n");
			//	printf("%08x%08x%08x%08x\n", rk0, rk1, rk2, rk3);
			//}
		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
			(t4S[(temp >>  8) & 0xff] & 0x00ff0000) ^
			(t4S[(temp      ) & 0xff] & 0x0000ff00) ^
			(t4S[(temp >> 24)       ] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t3) & 0xFF] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t0) & 0xFF] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t1) & 0xFF] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t2) & 0xFF] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table
// 1 Table -> arithmetic shift: 2 shift 1 and
__global__ void exhaustiveSearchWithOneTable(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE];
	__shared__ u32 t4S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t0S[threadIdx.x] = t0G[threadIdx.x];
		t4S[threadIdx.x] = t4G[threadIdx.x];

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	ull threadRangeStart = (ull)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
				(t4S[(temp >>  8) & 0xff] & 0x00ff0000) ^
				(t4S[(temp      ) & 0xff] & 0x0000ff00) ^
				(t4S[(temp >> 24)       ] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24] ^ arithmeticRightShift(t0S[(s1 >> 16) & 0xFF], 8) ^ arithmeticRightShift(t0S[(s2 >> 8) & 0xFF], 16) ^ arithmeticRightShift(t0S[s3 & 0xFF], 24) ^ rk0;
			t1 = t0S[s1 >> 24] ^ arithmeticRightShift(t0S[(s2 >> 16) & 0xFF], 8) ^ arithmeticRightShift(t0S[(s3 >> 8) & 0xFF], 16) ^ arithmeticRightShift(t0S[s0 & 0xFF], 24) ^ rk1;
			t2 = t0S[s2 >> 24] ^ arithmeticRightShift(t0S[(s3 >> 16) & 0xFF], 8) ^ arithmeticRightShift(t0S[(s0 >> 8) & 0xFF], 16) ^ arithmeticRightShift(t0S[s1 & 0xFF], 24) ^ rk2;
			t3 = t0S[s3 >> 24] ^ arithmeticRightShift(t0S[(s0 >> 16) & 0xFF], 8) ^ arithmeticRightShift(t0S[(s1 >> 8) & 0xFF], 16) ^ arithmeticRightShift(t0S[s2 & 0xFF], 24) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
			(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
			(t4S[(temp) & 0xff] & 0x0000ff00) ^
			(t4S[(temp >> 24)] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t3) & 0xFF] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t0) & 0xFF] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t1) & 0xFF] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t2) & 0xFF] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: 2 shift 1 and
__global__ void exhaustiveSearchWithOneTableExtendedSharedMemory(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t4S[threadIdx.x] = t4G[threadIdx.x];
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	ull threadRangeStart = (ull)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
				(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
				(t4S[(temp) & 0xff] & 0x0000ff00) ^
				(t4S[(temp >> 24)] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s3 & 0xFF][warpThreadIndex], 24) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s0 & 0xFF][warpThreadIndex], 24) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s1 & 0xFF][warpThreadIndex], 24) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s2 & 0xFF][warpThreadIndex], 24) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
			(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
			(t4S[(temp) & 0xff] & 0x0000ff00) ^
			(t4S[(temp >> 24)] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t3) & 0xFF] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t0) & 0xFF] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t1) & 0xFF] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t2) & 0xFF] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
__global__ void exhaustiveSearchWithOneTableExtendedSharedMemoryBytePerm(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t4S[threadIdx.x] = t4G[threadIdx.x];
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	ull threadRangeStart = (ull)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
				(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
				(t4S[(temp) & 0xff] & 0x0000ff00) ^
				(t4S[(temp >> 24)] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
			(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
			(t4S[(temp) & 0xff] & 0x0000ff00) ^
			(t4S[(temp >> 24)] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t3) & 0xFF] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t0) & 0xFF] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t1) & 0xFF] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t2) & 0xFF] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// SBox[256] is partly expanded
__global__ void exhaustiveSearchWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S_0[PARTLY_DIVIDE_THRESHOLD][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S_1[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < PARTLY_DIVIDE_THRESHOLD) {
			for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
				t4S_0[threadIdx.x][bankIndex] = t4G[threadIdx.x];
			}
		} else {
			t4S_1[threadIdx.x] = t4G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	ull threadRangeStart = (ull)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(returnPartlyExpandedTableResult(t4S_0, t4S_1, (temp >> 16) & 0xff, warpThreadIndex) & 0xff000000) ^
				(returnPartlyExpandedTableResult(t4S_0, t4S_1, (temp >>  8) & 0xff, warpThreadIndex) & 0x00ff0000) ^
				(returnPartlyExpandedTableResult(t4S_0, t4S_1, (temp      ) & 0xff, warpThreadIndex) & 0x0000ff00) ^
				(returnPartlyExpandedTableResult(t4S_0, t4S_1, (temp >> 24) & 0xff, warpThreadIndex) & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(returnPartlyExpandedTableResult(t4S_0, t4S_1, (temp >> 16) & 0xff, warpThreadIndex) & 0xff000000) ^
			(returnPartlyExpandedTableResult(t4S_0, t4S_1, (temp >>  8) & 0xff, warpThreadIndex) & 0x00ff0000) ^
			(returnPartlyExpandedTableResult(t4S_0, t4S_1, (temp      ) & 0xff, warpThreadIndex) & 0x0000ff00) ^
			(returnPartlyExpandedTableResult(t4S_0, t4S_1, (temp >> 24) & 0xff, warpThreadIndex) & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.

		s0 = (returnPartlyExpandedTableResult(t4S_0, t4S_1, t0 >> 24, warpThreadIndex) & 0xFF000000) ^ 
			(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t1 >> 16) & 0xff, warpThreadIndex) & 0x00FF0000) ^ 
			(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t2 >> 8) & 0xff, warpThreadIndex) & 0x0000FF00) ^ 
			(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t3) & 0xFF, warpThreadIndex) & 0x000000FF) ^ 
			rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (returnPartlyExpandedTableResult(t4S_0, t4S_1, t1 >> 24, warpThreadIndex) & 0xFF000000) ^ 
				(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t2 >> 16) & 0xff, warpThreadIndex) & 0x00FF0000) ^ 
				(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t3 >> 8) & 0xff, warpThreadIndex) & 0x0000FF00) ^
				(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t0) & 0xFF, warpThreadIndex) & 0x000000FF) ^
				rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (returnPartlyExpandedTableResult(t4S_0, t4S_1, t2 >> 24, warpThreadIndex) & 0xFF000000) ^ 
					(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t3 >> 16) & 0xff, warpThreadIndex) & 0x00FF0000) ^
					(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t0 >> 8) & 0xff, warpThreadIndex) & 0x0000FF00) ^
					(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t1) & 0xFF, warpThreadIndex) & 0x000000FF) ^
					rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (returnPartlyExpandedTableResult(t4S_0, t4S_1, t3 >> 24, warpThreadIndex) & 0xFF000000) ^ 
						(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t0 >> 16) & 0xff, warpThreadIndex) & 0x00FF0000) ^
						(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t1 >> 8) & 0xff, warpThreadIndex) & 0x0000FF00) ^
						(returnPartlyExpandedTableResult(t4S_0, t4S_1, (t2) & 0xFF, warpThreadIndex) & 0x000000FF) ^
						rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// 4 S-box, each shifted
__global__ void exhaustiveSearchWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4_0G, u32* t4_1G, u32* t4_2G, u32* t4_3G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4_0S[TABLE_SIZE];
	__shared__ u32 t4_1S[TABLE_SIZE];
	__shared__ u32 t4_2S[TABLE_SIZE];
	__shared__ u32 t4_3S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t4_0S[threadIdx.x] = t4_0G[threadIdx.x];
		t4_1S[threadIdx.x] = t4_1G[threadIdx.x];
		t4_2S[threadIdx.x] = t4_2G[threadIdx.x];
		t4_3S[threadIdx.x] = t4_3G[threadIdx.x];
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	ull threadRangeStart = (ull)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^ t4_3S[(temp >> 16) & 0xff] ^ t4_2S[(temp >> 8) & 0xff] ^ t4_1S[(temp) & 0xff] ^ t4_0S[(temp >> 24)] ^ rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^ t4_3S[(temp >> 16) & 0xff] ^ t4_2S[(temp >> 8) & 0xff] ^ t4_1S[(temp) & 0xff] ^ t4_0S[(temp >> 24)] ^ rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = t4_3S[t0 >> 24] ^ t4_2S[(t1 >> 16) & 0xff] ^ t4_1S[(t2 >> 8) & 0xff] ^ t4_0S[(t3) & 0xFF] ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = t4_3S[t1 >> 24] ^ t4_2S[(t2 >> 16) & 0xff] ^ t4_1S[(t3 >> 8) & 0xff] ^ t4_0S[(t0) & 0xFF] ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = t4_3S[t2 >> 24] ^ t4_2S[(t3 >> 16) & 0xff] ^ t4_1S[(t0 >> 8) & 0xff] ^ t4_0S[(t1) & 0xFF] ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = t4_3S[t3 >> 24] ^ t4_2S[(t0 >> 16) & 0xff] ^ t4_1S[(t1 >> 8) & 0xff] ^ t4_0S[(t2) & 0xFF] ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Key expansion from given key set, populate rk[44]
void keyExpansion(u32* key, u32* rk) {

	u32 rk0, rk1, rk2, rk3;
	rk0 = key[0];
	rk1 = key[1];
	rk2 = key[2];
	rk3 = key[3];

	rk[0] = rk0;
	rk[1] = rk1;
	rk[2] = rk2;
	rk[3] = rk3;

	for (u8 roundCount = 0; roundCount < ROUND_COUNT; roundCount++) {
		u32 temp = rk3;
		rk0 = rk0 ^ T4_3[(temp >> 16) & 0xff] ^ T4_2[(temp >> 8) & 0xff] ^ T4_1[(temp) & 0xff] ^ T4_0[(temp >> 24)] ^ RCON32[roundCount];
		rk1 = rk1 ^ rk0;
		rk2 = rk2 ^ rk1;
		rk3 = rk2 ^ rk3;

		rk[roundCount * 4 + 4] = rk0;
		rk[roundCount * 4 + 5] = rk1;
		rk[roundCount * 4 + 6] = rk2;
		rk[roundCount * 4 + 7] = rk3;
	}
}

// CTR encryption with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// 4 S-box, each shifted
__global__ void counterWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox(u32* pt, u32* rk, u32* t0G, u32* t4_0G, u32* t4_1G, u32* t4_2G, u32* t4_3G, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4_0S[TABLE_SIZE];
	__shared__ u32 t4_1S[TABLE_SIZE];
	__shared__ u32 t4_2S[TABLE_SIZE];
	__shared__ u32 t4_3S[TABLE_SIZE];
	__shared__ u32 rkS[TABLE_BASED_KEY_LIST_ROW_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t4_0S[threadIdx.x] = t4_0G[threadIdx.x];
		t4_1S[threadIdx.x] = t4_1G[threadIdx.x];
		t4_2S[threadIdx.x] = t4_2G[threadIdx.x];
		t4_3S[threadIdx.x] = t4_3G[threadIdx.x];
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < TABLE_BASED_KEY_LIST_ROW_SIZE) {
			rkS[threadIdx.x] = rk[threadIdx.x];
		}

	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	ull threadRangeStart = (ull)threadIndex * threadRange;
	pt2Init = pt2Init + threadRangeStart / MAX_U32;
	pt3Init = pt3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		if (threadIndex == 1048575 && rangeCount > (threadRange - 10)) {
			printf("Plaintext  : %08x %08x %08x %08x\n", s0, s1, s2, s3);
		}

		// First round just XORs input with key.
		s0 = s0 ^ rkS[0];
		s1 = s1 ^ rkS[1];
		s2 = s2 ^ rkS[2];
		s3 = s3 ^ rkS[3];

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Table based round function
			u32 rkStart = roundCount * 4 + 4;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 3];

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
		s0 = t4_3S[t0 >> 24] ^ t4_2S[(t1 >> 16) & 0xff] ^ t4_1S[(t2 >> 8) & 0xff] ^ t4_0S[(t3) & 0xFF] ^ rkS[40];
		s1 = t4_3S[t1 >> 24] ^ t4_2S[(t2 >> 16) & 0xff] ^ t4_1S[(t3 >> 8) & 0xff] ^ t4_0S[(t0) & 0xFF] ^ rkS[41];
		s2 = t4_3S[t2 >> 24] ^ t4_2S[(t3 >> 16) & 0xff] ^ t4_1S[(t0 >> 8) & 0xff] ^ t4_0S[(t1) & 0xFF] ^ rkS[42];
		s3 = t4_3S[t3 >> 24] ^ t4_2S[(t0 >> 16) & 0xff] ^ t4_1S[(t1 >> 8) & 0xff] ^ t4_0S[(t2) & 0xFF] ^ rkS[43];

		if (threadIndex == 1048575 && rangeCount > (threadRange - 10)) {
			printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s1, s3);
		}

		// Overflow
		if (pt3Init == MAX_U32) {
			pt2Init++;
		}

		// Create key as 32 bit unsigned integers
		pt3Init++;
	}
}

int main() {

	// Allocate key
	u32* rk;
	gpuErrorCheck(hipMallocManaged(&rk, 4 * sizeof(u32)));
	//rk[0] = 0x00000000U;
	//rk[1] = 0x00000000U;
	//rk[2] = 0x00000000U;
	//rk[3] = 0x00000000U;

	// aes-cipher-internals.xlsx
	rk[0] = 0x2B7E1516U;
	rk[1] = 0x28AED2A6U;
	rk[2] = 0xABF71588U;
	rk[3] = 0x09CF4F3CU;

	// Allocate plaintext
	u32* pt;
	gpuErrorCheck(hipMallocManaged(&pt, 4 * sizeof(u32)));
	pt[0] = 0x00000000U;
	pt[1] = 0x00000000U;
	pt[2] = 0x00000000U;
	pt[3] = 0x00000000U;

	// aes-cipher-internals.xlsx
	//pt[0] = 0x3243F6A8U;
	//pt[1] = 0x885A308DU;
	//pt[2] = 0x313198A2U;
	//pt[3] = 0xE0370734U;

	// Allocate ciphertext
	u32* ct;
	gpuErrorCheck(hipMallocManaged(&ct, 4 * sizeof(u32)));
	ct[0] = 0x4390c373U;
	ct[1] = 0xd11979acU;
	ct[2] = 0x6236104cU;
	ct[3] = 0xa3d85b88U;

	// aes-cipher-internals.xlsx
	//ct[0] = 0x3925841DU;
	//ct[1] = 0x02DC09FBU;
	//ct[2] = 0xDC118597U;
	//ct[3] = 0x196A0B32U;

	// CTR round keys
	u32* roundKeys;
	gpuErrorCheck(hipMallocManaged(&roundKeys, TABLE_BASED_KEY_LIST_ROW_SIZE * sizeof(u32)));

	// Allocate Tables
	u32 *t0, *t1, *t2, *t3, *t4, *t4_0, *t4_1, *t4_2, *t4_3;
	gpuErrorCheck(hipMallocManaged(&t0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_3, TABLE_SIZE * sizeof(u32)));
	for (int i = 0; i < TABLE_SIZE; i++) {
		t0[i] = T0[i];
		t1[i] = T1[i];
		t2[i] = T2[i];
		t3[i] = T3[i];
		t4[i] = T4[i];
		t4_0[i] = T4_0[i];
		t4_1[i] = T4_1[i];
		t4_2[i] = T4_2[i];
		t4_3[i] = T4_3[i];
	}

	// Allocate RCON values
	u32* rcon;
	gpuErrorCheck(hipMallocManaged(&rcon, RCON_SIZE * sizeof(u32)));
	for (int i = 0; i < RCON_SIZE; i++) {
		rcon[i] = RCON32[i];
	}

	// Calculate range
	u32* range;
	gpuErrorCheck(hipMallocManaged(&range, 1 * sizeof(u32)));

	int threadCount = BLOCKS * THREADS;
	
	double keyRange = pow(2, TWO_POWER_RANGE);
	double threadRange = keyRange / threadCount;
	*range = ceil(threadRange);

	// Printing info
	printf("------------------------------------\n");
	printf("Blocks                             : %d\n", BLOCKS);
	printf("Threads                            : %d\n", THREADS);
	printf("Total Thread count                 : %d\n", threadCount);
	printf("Key Range (power)                  : %d\n", TWO_POWER_RANGE);
	printf("Key Range (decimal)                : %.0f\n", keyRange);
	printf("Each Thread Key Range              : %.2f\n", threadRange);
	printf("Each Thread Key Range (kernel)     : %d\n", range[0]);
	printf("Total encryptions                  : %.0f\n", ceil(threadRange) * threadCount);
	printf("------------------------------------\n");
	printf("Initial Key                        : %08x %08x %08x %08x\n", rk[0], rk[1], rk[2], rk[3]);
	printf("Plaintext                          : %08x %08x %08x %08x\n", pt[0], pt[1], pt[2], pt[3]);
	printf("Ciphertext                         : %08x %08x %08x %08x\n", ct[0], ct[1], ct[2], ct[3]);
	printf("------------------------------------\n");

	clock_t beginTime = clock();

	//exhaustiveSearch<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t1, t2, t3, t4, rcon, range);

	//exhaustiveSearchWithOneTable<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t4, rcon, range);

	//exhaustiveSearchWithOneTableExtendedSharedMemory<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t4, rcon, range);

	//exhaustiveSearchWithOneTableExtendedSharedMemoryBytePerm<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t4, rcon, range);

	//exhaustiveSearchWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox<<<BLOCKS, THREADS >>>(pt, ct, rk, t0, t4, rcon, range);

	//exhaustiveSearchWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t4_0, t4_1, t4_2, t4_3, rcon, range);

	keyExpansion(rk, roundKeys);
	counterWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox<<<BLOCKS, THREADS>>>(pt, roundKeys, t0, t4_0, t4_1, t4_2, t4_3, range);

	hipDeviceSynchronize();
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);

	printLastCUDAError();

	// Printing info
	#ifdef  INFO
	printf("------------------------------------\n");
	u32 total;
	hipMemcpyFromSymbol(&total, HIP_SYMBOL(totalThreadCount), sizeof(u32));
	printf("Total Thread count                 : %d\n", total);
	hipMemcpyFromSymbol(&total, HIP_SYMBOL(maxThreadIndex), sizeof(u32));
	printf("Max Thread Index                   : %d\n", total);
	ulli totEncryption;
	hipMemcpyFromSymbol(&totEncryption, HIP_SYMBOL(totalEncryptions), sizeof(ulli));
	printf("Total encryptions                  : %lu\n", totEncryption);
	printf("------------------------------------\n");
	#endif // INFO

	// Free alocated arrays
	hipFree(rk);
	hipFree(pt);
	hipFree(ct);
	hipFree(t0);
	hipFree(t1);
	hipFree(t2);
	hipFree(t3);
	hipFree(t4);
	hipFree(rcon);
	hipFree(range);
	
	return 0;
}
