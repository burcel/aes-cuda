#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <ctime>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

#include <>
#include <hip/device_functions.h>

// Custom header 
#include "kernel.h"

//#define INFO 1
#ifdef  INFO
__device__ u32 totalThreadCount = 0;
__device__ u64 totalEncryptions = 0;
__device__ u32 maxThreadIndex = 0;
#endif // INFO

// Basic exhaustive search
// 4 Tables
__global__ void exhaustiveSearch(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t1G, u32* t2G, u32* t3G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE];
	__shared__ u32 t1S[TABLE_SIZE];
	__shared__ u32 t2S[TABLE_SIZE];
	__shared__ u32 t3S[TABLE_SIZE];
	__shared__ u32 t4S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t0S[threadIdx.x] = t0G[threadIdx.x];
		t1S[threadIdx.x] = t1G[threadIdx.x];
		t2S[threadIdx.x] = t2G[threadIdx.x];
		t3S[threadIdx.x] = t3G[threadIdx.x];
		t4S[threadIdx.x] = t4G[threadIdx.x];

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}
		 
		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		//if (threadIndex == 0 && rangeCount == 0) {
		//	printf("--Round: %d\n", 0);
		//	printf("%08x%08x%08x%08x\n", s0, s1, s2, s3);
		//	printf("-- Round Key\n");
		//	printf("%08x%08x%08x%08x\n", rk0, rk1, rk2, rk3);
		//}

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			// TODO: temp & 0xff000000
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
				(t4S[(temp >>  8) & 0xff] & 0x00ff0000) ^
				(t4S[(temp      ) & 0xff] & 0x0000ff00) ^
				(t4S[(temp >> 24)       ] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24] ^ t1S[(s1 >> 16) & 0xFF] ^ t2S[(s2 >> 8) & 0xFF] ^ t3S[s3 & 0xFF] ^ rk0;
			t1 = t0S[s1 >> 24] ^ t1S[(s2 >> 16) & 0xFF] ^ t2S[(s3 >> 8) & 0xFF] ^ t3S[s0 & 0xFF] ^ rk1;
			t2 = t0S[s2 >> 24] ^ t1S[(s3 >> 16) & 0xFF] ^ t2S[(s0 >> 8) & 0xFF] ^ t3S[s1 & 0xFF] ^ rk2;
			t3 = t0S[s3 >> 24] ^ t1S[(s0 >> 16) & 0xFF] ^ t2S[(s1 >> 8) & 0xFF] ^ t3S[s2 & 0xFF] ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

			//if (threadIndex == 0 && rangeCount == 0) {
			//	printf("--Round: %d\n", roundCount);
			//	printf("%08x%08x%08x%08x\n", s0, s1, s2, s3);
			//	printf("-- Round Key\n");
			//	printf("%08x%08x%08x%08x\n", rk0, rk1, rk2, rk3);
			//}
		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
			(t4S[(temp >>  8) & 0xff] & 0x00ff0000) ^
			(t4S[(temp      ) & 0xff] & 0x0000ff00) ^
			(t4S[(temp >> 24)       ] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t3) & 0xFF] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t0) & 0xFF] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t1) & 0xFF] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t2) & 0xFF] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table
// 1 Table -> arithmetic shift: 2 shift 1 and
__global__ void exhaustiveSearchWithOneTable(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE];
	__shared__ u32 t4S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t0S[threadIdx.x] = t0G[threadIdx.x];
		t4S[threadIdx.x] = t4G[threadIdx.x];

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
				(t4S[(temp >>  8) & 0xff] & 0x00ff0000) ^
				(t4S[(temp      ) & 0xff] & 0x0000ff00) ^
				(t4S[(temp >> 24)       ] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24] ^ arithmeticRightShift(t0S[(s1 >> 16) & 0xFF], 8) ^ arithmeticRightShift(t0S[(s2 >> 8) & 0xFF], 16) ^ arithmeticRightShift(t0S[s3 & 0xFF], 24) ^ rk0;
			t1 = t0S[s1 >> 24] ^ arithmeticRightShift(t0S[(s2 >> 16) & 0xFF], 8) ^ arithmeticRightShift(t0S[(s3 >> 8) & 0xFF], 16) ^ arithmeticRightShift(t0S[s0 & 0xFF], 24) ^ rk1;
			t2 = t0S[s2 >> 24] ^ arithmeticRightShift(t0S[(s3 >> 16) & 0xFF], 8) ^ arithmeticRightShift(t0S[(s0 >> 8) & 0xFF], 16) ^ arithmeticRightShift(t0S[s1 & 0xFF], 24) ^ rk2;
			t3 = t0S[s3 >> 24] ^ arithmeticRightShift(t0S[(s0 >> 16) & 0xFF], 8) ^ arithmeticRightShift(t0S[(s1 >> 8) & 0xFF], 16) ^ arithmeticRightShift(t0S[s2 & 0xFF], 24) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
			(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
			(t4S[(temp) & 0xff] & 0x0000ff00) ^
			(t4S[(temp >> 24)] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t3) & 0xFF] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t0) & 0xFF] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t1) & 0xFF] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t2) & 0xFF] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: 2 shift 1 and
__global__ void exhaustiveSearchWithOneTableExtendedSharedMemory(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t4S[threadIdx.x] = t4G[threadIdx.x];
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
				(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
				(t4S[(temp) & 0xff] & 0x0000ff00) ^
				(t4S[(temp >> 24)] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s3 & 0xFF][warpThreadIndex], 24) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s0 & 0xFF][warpThreadIndex], 24) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s1 & 0xFF][warpThreadIndex], 24) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShift(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], 8) ^ arithmeticRightShift(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], 16) ^ arithmeticRightShift(t0S[s2 & 0xFF][warpThreadIndex], 24) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
			(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
			(t4S[(temp) & 0xff] & 0x0000ff00) ^
			(t4S[(temp >> 24)] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t3) & 0xFF] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t0) & 0xFF] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t1) & 0xFF] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t2) & 0xFF] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
__global__ void exhaustiveSearchWithOneTableExtendedSharedMemoryBytePerm(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t4S[threadIdx.x] = t4G[threadIdx.x];
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
				(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
				(t4S[(temp) & 0xff] & 0x0000ff00) ^
				(t4S[(temp >> 24)] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff] & 0xff000000) ^
			(t4S[(temp >> 8) & 0xff] & 0x00ff0000) ^
			(t4S[(temp) & 0xff] & 0x0000ff00) ^
			(t4S[(temp >> 24)] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t3) & 0xFF] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t0) & 0xFF] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t1) & 0xFF] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff] & 0x0000FF00) ^ (t4S[(t2) & 0xFF] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// SBox[256] is partly expanded
__global__ void exhaustiveSearchWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {
			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(t4S[(temp >> 16) & 0xff][warpThreadIndexSBox] & 0xff000000) ^
				(t4S[(temp >>  8) & 0xff][warpThreadIndexSBox] & 0x00ff0000) ^
				(t4S[(temp      ) & 0xff][warpThreadIndexSBox] & 0x0000ff00) ^
				(t4S[(temp >> 24)       ][warpThreadIndexSBox] & 0x000000ff) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff][warpThreadIndexSBox] & 0xff000000) ^
			(t4S[(temp >>  8) & 0xff][warpThreadIndexSBox] & 0x00ff0000) ^
			(t4S[(temp      ) & 0xff][warpThreadIndexSBox] & 0x0000ff00) ^
			(t4S[(temp >> 24)       ][warpThreadIndexSBox] & 0x000000ff) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// 4 S-box, each shifted
__global__ void exhaustiveSearchWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4_0G, u32* t4_1G, u32* t4_2G, u32* t4_3G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4_0S[TABLE_SIZE];
	__shared__ u32 t4_1S[TABLE_SIZE];
	__shared__ u32 t4_2S[TABLE_SIZE];
	__shared__ u32 t4_3S[TABLE_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t4_0S[threadIdx.x] = t4_0G[threadIdx.x];
		t4_1S[threadIdx.x] = t4_1G[threadIdx.x];
		t4_2S[threadIdx.x] = t4_2G[threadIdx.x];
		t4_3S[threadIdx.x] = t4_3G[threadIdx.x];
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U32;
	rk3Init = rk3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^ t4_3S[(temp >> 16) & 0xff] ^ t4_2S[(temp >> 8) & 0xff] ^ t4_1S[(temp) & 0xff] ^ t4_0S[(temp >> 24)] ^ rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk0;
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk1;
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk2;
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^ t4_3S[(temp >> 16) & 0xff] ^ t4_2S[(temp >> 8) & 0xff] ^ t4_1S[(temp) & 0xff] ^ t4_0S[(temp >> 24)] ^ rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = t4_3S[t0 >> 24] ^ t4_2S[(t1 >> 16) & 0xff] ^ t4_1S[(t2 >> 8) & 0xff] ^ t4_0S[(t3) & 0xFF] ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = t4_3S[t1 >> 24] ^ t4_2S[(t2 >> 16) & 0xff] ^ t4_1S[(t3 >> 8) & 0xff] ^ t4_0S[(t0) & 0xFF] ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = t4_3S[t2 >> 24] ^ t4_2S[(t3 >> 16) & 0xff] ^ t4_1S[(t0 >> 8) & 0xff] ^ t4_0S[(t1) & 0xFF] ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = t4_3S[t3 >> 24] ^ t4_2S[(t0 >> 16) & 0xff] ^ t4_1S[(t1 >> 8) & 0xff] ^ t4_0S[(t2) & 0xFF] ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk3Init == MAX_U32) {
			rk2Init++;
		}

		// Create key as 32 bit unsigned integers
		rk3Init++;
	}
}

// Key expansion from given key set, populate rk[44]
void keyExpansion(u32* key, u32* rk) {

	u32 rk0, rk1, rk2, rk3;
	rk0 = key[0];
	rk1 = key[1];
	rk2 = key[2];
	rk3 = key[3];

	rk[0] = rk0;
	rk[1] = rk1;
	rk[2] = rk2;
	rk[3] = rk3;

	for (u8 roundCount = 0; roundCount < ROUND_COUNT; roundCount++) {
		u32 temp = rk3;
		rk0 = rk0 ^ T4_3[(temp >> 16) & 0xff] ^ T4_2[(temp >> 8) & 0xff] ^ T4_1[(temp) & 0xff] ^ T4_0[(temp >> 24)] ^ RCON32[roundCount];
		rk1 = rk1 ^ rk0;
		rk2 = rk2 ^ rk1;
		rk3 = rk2 ^ rk3;

		rk[roundCount * 4 + 4] = rk0;
		rk[roundCount * 4 + 5] = rk1;
		rk[roundCount * 4 + 6] = rk2;
		rk[roundCount * 4 + 7] = rk3;
	}
}

// CTR encryption with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// SBox[256] is partly expanded
__global__ void counterWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox(u32* pt, u32* rk, u32* t0G, u32* t4G, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rkS[TABLE_BASED_KEY_LIST_ROW_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {
			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];
		}

		if (threadIdx.x < TABLE_BASED_KEY_LIST_ROW_SIZE) {
			rkS[threadIdx.x] = rk[threadIdx.x];
		}

	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	u32 s0, s1, s2, s3;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	pt2Init = pt2Init + threadRangeStart / MAX_U32;
	pt3Init = pt3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		// Create plaintext as 32 bit unsigned integers
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rkS[0];
		s1 = s1 ^ rkS[1];
		s2 = s2 ^ rkS[2];
		s3 = s3 ^ rkS[3];

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Table based round function
			u32 rkStart = roundCount * 4 + 4;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 3];

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[40];
		s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[41];
		s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[42];
		s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[43];

		//if (threadIndex == 1048576) {
		//	printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
		//}

		// Overflow
		if (pt3Init == MAX_U32) {
			pt2Init++;
		}

		// Create key as 32 bit unsigned integers
		pt3Init++;
	}

	if (threadIndex == 1048575) {
		printf("Plaintext : %08x %08x %08x %08x\n", pt0Init, pt1Init, pt2Init, pt3Init);
		printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
	}

}

// CTR encryption with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// 4 S-box, each shifted
__global__ void counterWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox(u32* pt, u32* rk, u32* t0G, u32* t4_0G, u32* t4_1G, u32* t4_2G, u32* t4_3G, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4_0S[TABLE_SIZE];
	__shared__ u32 t4_1S[TABLE_SIZE];
	__shared__ u32 t4_2S[TABLE_SIZE];
	__shared__ u32 t4_3S[TABLE_SIZE];
	__shared__ u32 rkS[TABLE_BASED_KEY_LIST_ROW_SIZE];

	if (threadIdx.x < TABLE_SIZE) {
		t4_0S[threadIdx.x] = t4_0G[threadIdx.x];
		t4_1S[threadIdx.x] = t4_1G[threadIdx.x];
		t4_2S[threadIdx.x] = t4_2G[threadIdx.x];
		t4_3S[threadIdx.x] = t4_3G[threadIdx.x];
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		if (threadIdx.x < TABLE_BASED_KEY_LIST_ROW_SIZE) {
			rkS[threadIdx.x] = rk[threadIdx.x];
		}

	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	u32 s0, s1, s2, s3;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	pt2Init = pt2Init + threadRangeStart / MAX_U32;
	pt3Init = pt3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		// Create plaintext as 32 bit unsigned integers
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rkS[0];
		s1 = s1 ^ rkS[1];
		s2 = s2 ^ rkS[2];
		s3 = s3 ^ rkS[3];

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Table based round function
			u32 rkStart = roundCount * 4 + 4;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 3];

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
		s0 = t4_3S[t0 >> 24] ^ t4_2S[(t1 >> 16) & 0xff] ^ t4_1S[(t2 >> 8) & 0xff] ^ t4_0S[(t3) & 0xFF] ^ rkS[40];
		s1 = t4_3S[t1 >> 24] ^ t4_2S[(t2 >> 16) & 0xff] ^ t4_1S[(t3 >> 8) & 0xff] ^ t4_0S[(t0) & 0xFF] ^ rkS[41];
		s2 = t4_3S[t2 >> 24] ^ t4_2S[(t3 >> 16) & 0xff] ^ t4_1S[(t0 >> 8) & 0xff] ^ t4_0S[(t1) & 0xFF] ^ rkS[42];
		s3 = t4_3S[t3 >> 24] ^ t4_2S[(t0 >> 16) & 0xff] ^ t4_1S[(t1 >> 8) & 0xff] ^ t4_0S[(t2) & 0xFF] ^ rkS[43];

		//if (s0 == 0x00000000U) {
		//	printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s1, s3);
		//}

		// Overflow
		if (pt3Init == MAX_U32) {
			pt2Init++;
		}

		// Create key as 32 bit unsigned integers
		pt3Init++;
	}

	if (threadIndex == 1048575) {
		printf("Plaintext : %08x %08x %08x %08x\n", pt0Init, pt1Init, pt2Init, pt3Init);
		printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
	}
}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// SBox[256] is partly expanded
__global__ void exhaustiveSearch192WithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];


	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {
			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init, rk4Init, rk5Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];
	rk4Init = rk[4];
	rk5Init = rk[5];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk4Init = rk4Init + threadRangeStart / MAX_U32;
	rk5Init = rk5Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		// Calculate round keys
		u32 rk0, rk1, rk2, rk3, rk4, rk5;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;
		rk4 = rk4Init;
		rk5 = rk5Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		u8 rconIndex = 0;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1_192; roundCount++) {
			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT);
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT);
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT);
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT);

			// Add round key
			if (roundCount % 3 == 0) {
				t0 = t0 ^ rk4;
				t1 = t1 ^ rk5;
				// Calculate round key
				u32 temp = rk5;
				rk0 = rk0 ^
					(t4S[(temp >> 16) & 0xff][warpThreadIndexSBox] & 0xff000000) ^
					(t4S[(temp >>  8) & 0xff][warpThreadIndexSBox] & 0x00ff0000) ^
					(t4S[(temp      ) & 0xff][warpThreadIndexSBox] & 0x0000ff00) ^
					(t4S[(temp >> 24)       ][warpThreadIndexSBox] & 0x000000ff) ^
					rconS[rconIndex++];
				rk1 = rk1 ^ rk0;
				rk2 = rk2 ^ rk1;
				rk3 = rk3 ^ rk2;
				rk4 = rk4 ^ rk3;
				rk5 = rk5 ^ rk4;

				t2 = t2 ^ rk0;
				t3 = t3 ^ rk1;
			} else if (roundCount % 3 == 1) {
				t0 = t0 ^ rk2;
				t1 = t1 ^ rk3;
				t2 = t2 ^ rk4;
				t3 = t3 ^ rk5;
			} else {
				// Calculate round key
				u32 temp = rk5;
				rk0 = rk0 ^
					(t4S[(temp >> 16) & 0xff][warpThreadIndexSBox] & 0xff000000) ^
					(t4S[(temp >>  8) & 0xff][warpThreadIndexSBox] & 0x00ff0000) ^
					(t4S[(temp      ) & 0xff][warpThreadIndexSBox] & 0x0000ff00) ^
					(t4S[(temp >> 24)       ][warpThreadIndexSBox] & 0x000000ff) ^
					rconS[rconIndex++];
				rk1 = rk1 ^ rk0;
				rk2 = rk2 ^ rk1;
				rk3 = rk3 ^ rk2;
				rk4 = rk4 ^ rk3;
				rk5 = rk5 ^ rk4;

				t0 = t0 ^ rk0;
				t1 = t1 ^ rk1;
				t2 = t2 ^ rk2;
				t3 = t3 ^ rk3;
			}

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;
		}

		// Calculate the last round key
		u32 temp = rk5;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff][warpThreadIndexSBox] & 0xff000000) ^
			(t4S[(temp >> 8) & 0xff][warpThreadIndexSBox] & 0x00ff0000) ^
			(t4S[(temp) & 0xff][warpThreadIndexSBox] & 0x0000ff00) ^
			(t4S[(temp >> 24)][warpThreadIndexSBox] & 0x000000ff) ^
			rconS[rconIndex];

		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk5Init == MAX_U32) {
			rk4Init++;
		}

		// Create key as 32 bit unsigned integers
		rk5Init++;
	}
}

// Key expansion from given key set, populate rk[52]
void keyExpansion192(u32* key, u32* rk) {

	u32 rk0, rk1, rk2, rk3, rk4, rk5;
	rk0 = key[0];
	rk1 = key[1];
	rk2 = key[2];
	rk3 = key[3];
	rk4 = key[4];
	rk5 = key[5];

	rk[0] = rk0;
	rk[1] = rk1;
	rk[2] = rk2;
	rk[3] = rk3;
	rk[4] = rk4;
	rk[5] = rk5;

	for (u8 roundCount = 0; roundCount < ROUND_COUNT_192; roundCount++) {
		u32 temp = rk5;
		rk0 = rk0 ^ T4_3[(temp >> 16) & 0xff] ^ T4_2[(temp >> 8) & 0xff] ^ T4_1[(temp) & 0xff] ^ T4_0[(temp >> 24)] ^ RCON32[roundCount];
		rk1 = rk1 ^ rk0;
		rk2 = rk2 ^ rk1;
		rk3 = rk3 ^ rk2;
		rk4 = rk4 ^ rk3;
		rk5 = rk5 ^ rk4;

		rk[roundCount * 6 + 6] = rk0;
		rk[roundCount * 6 + 7] = rk1;
		rk[roundCount * 6 + 8] = rk2;
		rk[roundCount * 6 + 9] = rk3;
		if (roundCount == 7) {
			break;
		}
		rk[roundCount * 6 + 10] = rk4;
		rk[roundCount * 6 + 11] = rk5;
	}

	// Print keys
	//for (int i = 0;i < 52;i++) {
	//	printf("%08x ", rk[i]);
	//	if ((i+1) % 4 == 0) {
	//		printf("Round: %d\n", i / 4);
	//	}
	//}
}

// CTR encryption with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// SBox[256] is partly expanded
__global__ void counter192WithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox(u32* pt, u32* rk, u32* t0G, u32* t4G, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rkS[TABLE_BASED_KEY_LIST_SIZE_192];

	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {
			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];
		}

		if (threadIdx.x < TABLE_BASED_KEY_LIST_SIZE_192) {
			rkS[threadIdx.x] = rk[threadIdx.x];
		}

	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	u32 s0, s1, s2, s3;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	pt2Init = pt2Init + threadRangeStart / MAX_U32;
	pt3Init = pt3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		// Create plaintext as 32 bit unsigned integers
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rkS[0];
		s1 = s1 ^ rkS[1];
		s2 = s2 ^ rkS[2];
		s3 = s3 ^ rkS[3];

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1_192; roundCount++) {

			// Table based round function
			u32 rkStart = roundCount * 4 + 4;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 3];

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[48];
		s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[49];
		s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[50];
		s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[51];

		/*if (threadIndex == 0 && rangeCount == 0) {
			printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
		}*/

		// Overflow
		if (pt3Init == MAX_U32) {
			pt2Init++;
		}

		// Create key as 32 bit unsigned integers
		pt3Init++;
	}

	if (threadIndex == 1048575) {
		printf("Plaintext : %08x %08x %08x %08x\n", pt0Init, pt1Init, pt2Init, pt3Init);
		printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
	}

}

// Exhaustive search with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// SBox[256] is partly expanded
__global__ void exhaustiveSearch256WithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];


	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {
			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];
		}

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];
		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init, rk4Init, rk5Init, rk6Init, rk7Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];
	rk4Init = rk[4];
	rk5Init = rk[5];
	rk6Init = rk[6];
	rk7Init = rk[7];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk6Init = rk6Init + threadRangeStart / MAX_U32;
	rk7Init = rk7Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		// Calculate round keys
		u32 rk0, rk1, rk2, rk3, rk4, rk5, rk6, rk7;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;
		rk4 = rk4Init;
		rk5 = rk5Init;
		rk6 = rk6Init;
		rk7 = rk7Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		u8 rconIndex = 0;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1_256; roundCount++) {
			// Table based round function
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT);
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT);
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT);
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT);

			// Add round key
			if (roundCount % 2 == 0) {
				t0 = t0 ^ rk4;
				t1 = t1 ^ rk5;
				t2 = t2 ^ rk6;
				t3 = t3 ^ rk7;
			} else {
				// Calculate round key
				u32 temp = rk7;
				rk0 = rk0 ^
					(t4S[(temp >> 16) & 0xff][warpThreadIndexSBox] & 0xff000000) ^
					(t4S[(temp >> 8) & 0xff][warpThreadIndexSBox] & 0x00ff0000) ^
					(t4S[(temp) & 0xff][warpThreadIndexSBox] & 0x0000ff00) ^
					(t4S[(temp >> 24)][warpThreadIndexSBox] & 0x000000ff) ^
					rconS[rconIndex++];
				rk1 = rk1 ^ rk0;
				rk2 = rk2 ^ rk1;
				rk3 = rk3 ^ rk2;
				rk4 = rk4 ^
					(t4S[(rk3 >> 24) & 0xff][warpThreadIndexSBox] & 0xff000000) ^
					(t4S[(rk3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00ff0000) ^
					(t4S[(rk3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000ff00) ^
					(t4S[(rk3) & 0xff][warpThreadIndexSBox] & 0x000000ff);
				rk5 = rk5 ^ rk4;
				rk6 = rk6 ^ rk5;
				rk7 = rk7 ^ rk6;

				t0 = t0 ^ rk0;
				t1 = t1 ^ rk1;
				t2 = t2 ^ rk2;
				t3 = t3 ^ rk3;
			}

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;
		}

		// Calculate the last round key
		u32 temp = rk7;
		rk0 = rk0 ^
			(t4S[(temp >> 16) & 0xff][warpThreadIndexSBox] & 0xff000000) ^
			(t4S[(temp >> 8) & 0xff][warpThreadIndexSBox] & 0x00ff0000) ^
			(t4S[(temp) & 0xff][warpThreadIndexSBox] & 0x0000ff00) ^
			(t4S[(temp >> 24)][warpThreadIndexSBox] & 0x000000ff) ^
			rconS[rconIndex++];

		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key : %08x %08x %08x %08x\n", rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		// Overflow
		if (rk7Init == MAX_U32) {
			rk6Init++;
		}

		// Create key as 32 bit unsigned integers
		rk7Init++;
	}
}

// Key expansion from given key set, populate rk[52]
void keyExpansion256(u32* key, u32* rk) {

	u32 rk0, rk1, rk2, rk3, rk4, rk5, rk6, rk7;
	rk0 = key[0];
	rk1 = key[1];
	rk2 = key[2];
	rk3 = key[3];
	rk4 = key[4];
	rk5 = key[5];
	rk6 = key[6];
	rk7 = key[7];

	rk[0] = rk0;
	rk[1] = rk1;
	rk[2] = rk2;
	rk[3] = rk3;
	rk[4] = rk4;
	rk[5] = rk5;
	rk[6] = rk6;
	rk[7] = rk7;

	for (u8 roundCount = 0; roundCount < ROUND_COUNT_256; roundCount++) {
		u32 temp = rk7;
		rk0 = rk0 ^ T4_3[(temp >> 16) & 0xff] ^ T4_2[(temp >> 8) & 0xff] ^ T4_1[(temp) & 0xff] ^ T4_0[(temp >> 24)] ^ RCON32[roundCount];
		rk1 = rk1 ^ rk0;
		rk2 = rk2 ^ rk1;
		rk3 = rk3 ^ rk2;
		rk4 = rk4 ^ T4_3[(rk3 >> 24) & 0xff] ^ T4_2[(rk3 >> 16) & 0xff] ^ T4_1[(rk3 >> 8) & 0xff] ^ T4_0[rk3 & 0xff];
		rk5 = rk5 ^ rk4;
		rk6 = rk6 ^ rk5;
		rk7 = rk7 ^ rk6;

		rk[roundCount * 8 + 8] = rk0;
		rk[roundCount * 8 + 9] = rk1;
		rk[roundCount * 8 + 10] = rk2;
		rk[roundCount * 8 + 11] = rk3;
		if (roundCount == 6) {
			break;
		}
		rk[roundCount * 8 + 12] = rk4;
		rk[roundCount * 8 + 13] = rk5;
		rk[roundCount * 8 + 14] = rk6;
		rk[roundCount * 8 + 15] = rk7;
		
	}

	//for (int i = 0; i < 60; i++) {
	//	printf("%08x ", rk[i]);
	//	if ((i + 1) % 4 == 0) {
	//		printf("Round: %d\n", i / 4);
	//	}
	//}
}

// CTR encryption with one table extended as 32 columns
// 1 Table [256][32] -> arithmetic shift: __byte_perm function
// SBox[256] is partly expanded
__global__ void counter256WithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox(u32* pt, u32* rk, u32* t0G, u32* t4G, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;

	// <SHARED MEMORY>
	__shared__ u32 t0S[TABLE_SIZE][SHARED_MEM_BANK_SIZE];
	__shared__ u32 t4S[TABLE_SIZE][S_BOX_BANK_SIZE];
	__shared__ u32 rkS[TABLE_BASED_KEY_LIST_SIZE_256];

	if (threadIdx.x < TABLE_SIZE) {
		for (u8 bankIndex = 0; bankIndex < SHARED_MEM_BANK_SIZE; bankIndex++) {
			t0S[threadIdx.x][bankIndex] = t0G[threadIdx.x];
		}

		for (u8 bankIndex = 0; bankIndex < S_BOX_BANK_SIZE; bankIndex++) {
			t4S[threadIdx.x][bankIndex] = t4G[threadIdx.x];
		}

		if (threadIdx.x < TABLE_BASED_KEY_LIST_SIZE_256) {
			rkS[threadIdx.x] = rk[threadIdx.x];
		}

	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	u32 s0, s1, s2, s3;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	pt2Init = pt2Init + threadRangeStart / MAX_U32;
	pt3Init = pt3Init + threadRangeStart % MAX_U32;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		// Create plaintext as 32 bit unsigned integers
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rkS[0];
		s1 = s1 ^ rkS[1];
		s2 = s2 ^ rkS[2];
		s3 = s3 ^ rkS[3];

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1_256; roundCount++) {

			// Table based round function
			u32 rkStart = roundCount * 4 + 4;
			t0 = t0S[s0 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart];
			t1 = t0S[s1 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 1];
			t2 = t0S[s2 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 2];
			t3 = t0S[s3 >> 24][warpThreadIndex] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 16) & 0xFF][warpThreadIndex], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xFF][warpThreadIndex], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xFF][warpThreadIndex], SHIFT_3_RIGHT) ^ rkS[rkStart + 3];

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t1 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t2 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t3) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[56];
		s1 = (t4S[t1 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t2 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t3 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t0) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[57];
		s2 = (t4S[t2 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t3 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t0 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t1) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[58];
		s3 = (t4S[t3 >> 24][warpThreadIndexSBox] & 0xFF000000) ^ (t4S[(t0 >> 16) & 0xff][warpThreadIndexSBox] & 0x00FF0000) ^ (t4S[(t1 >> 8) & 0xff][warpThreadIndexSBox] & 0x0000FF00) ^ (t4S[(t2) & 0xFF][warpThreadIndexSBox] & 0x000000FF) ^ rkS[59];

		//if (threadIndex == 0 && rangeCount == 0) {
		//printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
		//}

		// Overflow
		if (pt3Init == MAX_U32) {
			pt2Init++;
		}

		// Create key as 32 bit unsigned integers
		pt3Init++;
	}

	if (threadIndex == 1048575) {
		printf("Plaintext : %08x %08x %08x %08x\n", pt0Init, pt1Init, pt2Init, pt3Init);
		printf("Ciphertext : %08x %08x %08x %08x\n", s0, s1, s2, s3);
	}

}


__global__ void smallAesExhaustiveSearch(u32* pt, u32* ct, u32* rk, u32* t0G, u32* t4G, u32* rconG, u32* range) {

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int warpThreadIndex = threadIdx.x & 31;
	int warpThreadIndexSBox = warpThreadIndex % S_BOX_BANK_SIZE;

	// <SHARED MEMORY>
	__shared__ u32 t0S[16];
	__shared__ u32 t4S[16];
	__shared__ u32 rconS[RCON_SIZE];
	__shared__ u32 ctS[U32_SIZE];

	if (threadIdx.x < 16) {
		t0S[threadIdx.x] = t0G[threadIdx.x];
		t4S[threadIdx.x] = t4G[threadIdx.x];

		if (threadIdx.x < RCON_SIZE) {
			rconS[threadIdx.x] = rconG[threadIdx.x];
		}

		if (threadIdx.x < U32_SIZE) {
			ctS[threadIdx.x] = ct[threadIdx.x];

		}
	}
	// </SHARED MEMORY>

	#ifdef  INFO
	atomicAdd(&totalThreadCount, 1);
	atomicMax(&maxThreadIndex, threadIndex);
	#endif // INFO

	// Wait until every thread is ready
	__syncthreads();

	u32 rk0Init, rk1Init, rk2Init, rk3Init;
	rk0Init = rk[0];
	rk1Init = rk[1];
	rk2Init = rk[2];
	rk3Init = rk[3];

	u32 pt0Init, pt1Init, pt2Init, pt3Init;
	pt0Init = pt[0];
	pt1Init = pt[1];
	pt2Init = pt[2];
	pt3Init = pt[3];

	u32 threadRange = *range;
	u64 threadRangeStart = (u64)threadIndex * threadRange;
	rk2Init = rk2Init + threadRangeStart / MAX_U16;
	rk3Init = rk3Init + threadRangeStart % MAX_U16;

	for (u32 rangeCount = 0; rangeCount < threadRange; rangeCount++) {

		#ifdef  INFO
		atomicAdd(&totalEncryptions, 1);
		#endif // INFO

		u32 rk0, rk1, rk2, rk3;
		rk0 = rk0Init;
		rk1 = rk1Init;
		rk2 = rk2Init;
		rk3 = rk3Init;

		// Create plaintext as 32 bit unsigned integers
		u32 s0, s1, s2, s3;
		s0 = pt0Init;
		s1 = pt1Init;
		s2 = pt2Init;
		s3 = pt3Init;

		// First round just XORs input with key.
		s0 = s0 ^ rk0;
		s1 = s1 ^ rk1;
		s2 = s2 ^ rk2;
		s3 = s3 ^ rk3;

		u32 t0, t1, t2, t3;
		for (u8 roundCount = 0; roundCount < ROUND_COUNT_MIN_1; roundCount++) {

			// Calculate round key
			u32 temp = rk3;
			rk0 = rk0 ^
				(t4S[(temp >>  8) & 0xf] & 0xf000) ^
				(t4S[(temp >>  4) & 0xf] & 0x0f00) ^
				(t4S[(temp      ) & 0xf] & 0x00f0) ^
				(t4S[(temp >> 12)      ] & 0x000f) ^
				rconS[roundCount];
			rk1 = rk1 ^ rk0;
			rk2 = rk2 ^ rk1;
			rk3 = rk2 ^ rk3;

			// Table based round function
			t0 = t0S[s0 >> 12] ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 8) & 0xF], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 4) & 0xF], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s3 & 0xF], SHIFT_3_RIGHT) ^ rk0;
			t1 = t0S[s1 >> 12] ^ arithmeticRightShiftBytePerm(t0S[(s2 >> 8) & 0xF], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 4) & 0xF], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s0 & 0xF], SHIFT_3_RIGHT) ^ rk1;
			t2 = t0S[s2 >> 12] ^ arithmeticRightShiftBytePerm(t0S[(s3 >> 8) & 0xF], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 4) & 0xF], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s1 & 0xF], SHIFT_3_RIGHT) ^ rk2;
			t3 = t0S[s3 >> 12] ^ arithmeticRightShiftBytePerm(t0S[(s0 >> 8) & 0xF], SHIFT_1_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[(s1 >> 4) & 0xF], SHIFT_2_RIGHT) ^ arithmeticRightShiftBytePerm(t0S[s2 & 0xF], SHIFT_3_RIGHT) ^ rk3;

			s0 = t0;
			s1 = t1;
			s2 = t2;
			s3 = t3;

		}

		// Calculate the last round key
		u32 temp = rk3;
		rk0 = rk0 ^
			(t4S[(temp >>  8) & 0xf] & 0xf000) ^
			(t4S[(temp >>  4) & 0xf] & 0x0f00) ^
			(t4S[(temp      ) & 0xf] & 0x00f0) ^
			(t4S[(temp >> 12)      ] & 0x000f) ^
			rconS[ROUND_COUNT_MIN_1];
		// Last round uses s-box directly and XORs to produce output.
		s0 = (t4S[t0 >> 12] & 0xF000) ^ (t4S[(t1 >> 8) & 0xf] & 0x0F00) ^ (t4S[(t2 >> 4) & 0xf] & 0x00F0) ^ (t4S[(t3) & 0xF] & 0x000F) ^ rk0;
		if (s0 == ctS[0]) {
			rk1 = rk1 ^ rk0;
			s1 = (t4S[t1 >> 12] & 0xF000) ^ (t4S[(t2 >> 8) & 0xf] & 0x0F00) ^ (t4S[(t3 >> 4) & 0xf] & 0x00F0) ^ (t4S[(t0) & 0xF] & 0x000F) ^ rk1;
			if (s1 == ctS[1]) {
				rk2 = rk2 ^ rk1;
				s2 = (t4S[t2 >> 12] & 0xF000) ^ (t4S[(t3 >> 8) & 0xf] & 0x0F00) ^ (t4S[(t0 >> 4) & 0xf] & 0x00F0) ^ (t4S[(t1) & 0xF] & 0x000F) ^ rk2;
				if (s2 == ctS[2]) {
					rk3 = rk2 ^ rk3;
					s3 = (t4S[t3 >> 12] & 0xF000) ^ (t4S[(t0 >> 8) & 0xf] & 0x0F00) ^ (t4S[(t1 >> 4) & 0xf] & 0x00F0) ^ (t4S[(t2) & 0xF] & 0x000F) ^ rk3;
					if (s3 == ctS[3]) {
						printf("! Found key %d : \n", threadIndex, rk0Init, rk1Init, rk2Init, rk3Init);
					}
				}
			}
		}

		if (threadIndex == 0) {
			printf("! CT %d : %08x %08x %08x %08x\n", threadIndex, s0, s1, s2, s3);
		}

		// Overflow
		if (rk3Init == MAX_U16) {
			rk2Init++;
			rk3Init = 0xFFFFFFFF;
		}

		rk3Init++;
	}
}

int main() {

	// <AES-128>
	// Allocate key
	u32* rk;
	gpuErrorCheck(hipMallocManaged(&rk, 4 * sizeof(u32)));
	rk[0] = 0x00000000U;
	rk[1] = 0x00000000U;
	rk[2] = 0x00000000U;
	rk[3] = 0x00000000U;

	// aes-cipher-internals.xlsx
	//rk[0] = 0x2B7E1516U;
	//rk[1] = 0x28AED2A6U;
	//rk[2] = 0xABF71588U;
	//rk[3] = 0x09CF4F3CU;

	// Allocate plaintext
	u32* pt;
	gpuErrorCheck(hipMallocManaged(&pt, 4 * sizeof(u32)));
	//pt[0] = 0x00000000U;
	//pt[1] = 0x00000000U;
	//pt[2] = 0x00000000U;
	//pt[3] = 0x00000000U;

	// aes-cipher-internals.xlsx
	pt[0] = 0x6bc1bee2U;
	pt[1] = 0x2e409f96U;
	pt[2] = 0xe93d7e11U;
	pt[3] = 0x7393172aU;

	// Allocate ciphertext
	u32* ct;
	gpuErrorCheck(hipMallocManaged(&ct, 4 * sizeof(u32)));
	ct[0] = 0xF3EED1BDU;
	ct[1] = 0xB5D2A03CU;
	ct[2] = 0x064B5A7EU;
	ct[3] = 0x3DB181F8U;

	// aes-cipher-internals.xlsx
	//ct[0] = 0x3925841DU;
	//ct[1] = 0x02DC09FBU;
	//ct[2] = 0xDC118597U;
	//ct[3] = 0x196A0B32U;

	// CTR round keys
	u32* roundKeys;
	gpuErrorCheck(hipMallocManaged(&roundKeys, TABLE_BASED_KEY_LIST_ROW_SIZE * sizeof(u32)));
	// </AES-128>

	// <AES-192>
	u32* rk192;
	gpuErrorCheck(hipMallocManaged(&rk192, 6 * sizeof(u32)));
	rk192[0] = 0x8e73b0f7U;
	rk192[1] = 0xda0e6452U;
	rk192[2] = 0xc810f32bU;
	rk192[3] = 0x809079e5U;
	rk192[4] = 0x62f8ead2U;
	rk192[5] = 0x522c6b7bU;
	// CTR round keys
	u32* roundKeys192;
	gpuErrorCheck(hipMallocManaged(&roundKeys192, TABLE_BASED_KEY_LIST_SIZE_192 * sizeof(u32)));
	// </AES-192>

	// <AES-256>
	u32* rk256;
	gpuErrorCheck(hipMallocManaged(&rk256, 8 * sizeof(u32)));
	rk256[0] = 0x603deb10U;
	rk256[1] = 0x15ca71beU;
	rk256[2] = 0x2b73aef0U;
	rk256[3] = 0x857d7781U;
	rk256[4] = 0x1f352c07U;
	rk256[5] = 0x3b6108d7U;
	rk256[6] = 0x2d9810a3U;
	rk256[7] = 0x0914dff4U;
	// CTR round keys
	u32* roundKeys256;
	gpuErrorCheck(hipMallocManaged(&roundKeys256, TABLE_BASED_KEY_LIST_SIZE_256 * sizeof(u32)));
	// </AES-192>

	// Allocate Tables
	u32 *t0, *t1, *t2, *t3, *t4, *t4_0, *t4_1, *t4_2, *t4_3;
	gpuErrorCheck(hipMallocManaged(&t0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t3, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_0, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_1, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_2, TABLE_SIZE * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4_3, TABLE_SIZE * sizeof(u32)));
	for (int i = 0; i < TABLE_SIZE; i++) {
		t0[i] = T0[i];
		t1[i] = T1[i];
		t2[i] = T2[i];
		t3[i] = T3[i];
		t4[i] = T4[i];
		t4_0[i] = T4_0[i];
		t4_1[i] = T4_1[i];
		t4_2[i] = T4_2[i];
		t4_3[i] = T4_3[i];
	}

	// Allocate RCON values
	u32* rcon;
	gpuErrorCheck(hipMallocManaged(&rcon, RCON_SIZE * sizeof(u32)));
	for (int i = 0; i < RCON_SIZE; i++) {
		rcon[i] = RCON32[i];
	}

	// -- Small AES --
	rk[0] = 0x00000000U;
	rk[1] = 0x00000000U;
	rk[2] = 0x00000000U;
	rk[3] = 0x0000FFFFU;

	pt[0] = 0x00006cbeU;
	pt[1] = 0x00002e40U;
	pt[2] = 0x0000e93dU;
	pt[3] = 0x00007393U;

	ct[0] = 0x00002ec7U;
	ct[1] = 0x000065c7U;
	ct[2] = 0x00005adcU;
	ct[3] = 0x000094e6U;

	u32 *t0Sml, *t1Sml, *t2Sml, *t3Sml, *t4Sml;
	gpuErrorCheck(hipMallocManaged(&t0Sml, 16 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t1Sml, 16 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t2Sml, 16 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t3Sml, 16 * sizeof(u32)));
	gpuErrorCheck(hipMallocManaged(&t4Sml, 16 * sizeof(u32)));
	for (int i = 0; i < 16; i++) {
		t0Sml[i] = T0_SML[i];
		t1Sml[i] = T1_SML[i];
		t2Sml[i] = T2_SML[i];
		t3Sml[i] = T3_SML[i];
		t4Sml[i] = T4_SML[i];
	}

	u32* rconSml;
	gpuErrorCheck(hipMallocManaged(&rconSml, RCON_SIZE * sizeof(u32)));
	for (int i = 0; i < RCON_SIZE; i++) {
		rconSml[i] = RCON_SML[i];
	}

	// Calculate range
	u32* range;
	gpuErrorCheck(hipMallocManaged(&range, 1 * sizeof(u32)));

	int threadCount = BLOCKS * THREADS;
	
	double keyRange = pow(2, TWO_POWER_RANGE);
	double threadRange = keyRange / threadCount;
	*range = ceil(threadRange);

	// Printing info
	printf("------------------------------------\n");
	printf("Blocks                             : %d\n", BLOCKS);
	printf("Threads                            : %d\n", THREADS);
	printf("Total Thread count                 : %d\n", threadCount);
	printf("Key Range (power)                  : %d\n", TWO_POWER_RANGE);
	printf("Key Range (decimal)                : %.0f\n", keyRange);
	printf("Each Thread Key Range              : %.2f\n", threadRange);
	printf("Each Thread Key Range (kernel)     : %d\n", range[0]);
	printf("Total encryptions                  : %.0f\n", ceil(threadRange) * threadCount);
	printf("------------------------------------\n");
	printf("Initial Key                        : %08x %08x %08x %08x\n", rk[0], rk[1], rk[2], rk[3]);
	printf("Plaintext                          : %08x %08x %08x %08x\n", pt[0], pt[1], pt[2], pt[3]);
	printf("Ciphertext                         : %08x %08x %08x %08x\n", ct[0], ct[1], ct[2], ct[3]);
	printf("------------------------------------\n");

	clock_t beginTime = clock();

	// -- AES-128 Exhaustive Search --

	//exhaustiveSearch<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t1, t2, t3, t4, rcon, range);

	//exhaustiveSearchWithOneTable<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t4, rcon, range);

	//exhaustiveSearchWithOneTableExtendedSharedMemory<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t4, rcon, range);

	//exhaustiveSearchWithOneTableExtendedSharedMemoryBytePerm<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t4, rcon, range);

	//exhaustiveSearchWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox<<<BLOCKS, THREADS >>>(pt, ct, rk, t0, t4, rcon, range);

	//exhaustiveSearchWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox<<<BLOCKS, THREADS>>>(pt, ct, rk, t0, t4_0, t4_1, t4_2, t4_3, rcon, range);

	// -- AES-128 CTR --

	//keyExpansion(rk, roundKeys);

	//counterWithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox<<<BLOCKS, THREADS>>>(pt, roundKeys, t0, t4, range);

	//counterWithOneTableExtendedSharedMemoryBytePerm4ShiftedSbox<<<BLOCKS, THREADS>>>(pt, roundKeys, t0, t4_0, t4_1, t4_2, t4_3, range);

	// -- AES-192 Exhaustive Search --
	//exhaustiveSearch192WithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox<<<BLOCKS, THREADS>>>(pt, ct, rk192, t0, t4, rcon, range);

	// -- AES-192 CTR --
	//keyExpansion192(rk192, roundKeys192);

	//counter192WithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox<<<BLOCKS, THREADS>>>(pt, roundKeys192, t0, t4, range);

	// -- AES-256 Exhaustive Search --
	//exhaustiveSearch256WithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox<<<BLOCKS, THREADS>>>(pt, ct, rk256, t0, t4, rcon, range);

	// -- AES-256 CTR --
	//keyExpansion256(rk256, roundKeys256);

	//counter256WithOneTableExtendedSharedMemoryBytePermPartlyExtendedSBox<<<BLOCKS, THREADS>>>(pt, roundKeys256, t0, t4, range);

	// -- Small AES --

	smallAesExhaustiveSearch<<<BLOCKS, THREADS>>>(pt, ct, rk, t0Sml, t4Sml, rconSml, range);

	hipDeviceSynchronize();
	printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);

	printLastCUDAError();

	// Printing info
	#ifdef  INFO
	printf("------------------------------------\n");
	u32 total;
	hipMemcpyFromSymbol(&total, HIP_SYMBOL(totalThreadCount), sizeof(u32));
	printf("Total Thread count                 : %d\n", total);
	hipMemcpyFromSymbol(&total, HIP_SYMBOL(maxThreadIndex), sizeof(u32));
	printf("Max Thread Index                   : %d\n", total);
	u64 totEncryption;
	hipMemcpyFromSymbol(&totEncryption, HIP_SYMBOL(totalEncryptions), sizeof(u64));
	printf("Total encryptions                  : %lu\n", totEncryption);
	printf("------------------------------------\n");
	#endif // INFO

	// Free alocated arrays
	hipFree(rk);
	hipFree(pt);
	hipFree(ct);
	hipFree(t0);
	hipFree(t1);
	hipFree(t2);
	hipFree(t3);
	hipFree(t4);
	hipFree(t4_0);
	hipFree(t4_1);
	hipFree(t4_2);
	hipFree(t4_3);
	hipFree(rcon);
	hipFree(range);
	hipFree(t0Sml);
	hipFree(t1Sml);
	hipFree(t2Sml);
	hipFree(t3Sml);
	hipFree(t4Sml);
	hipFree(rconSml);
	
	return 0;
}
