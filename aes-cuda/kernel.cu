// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <helper_cuda.h>


#include <hip/device_functions.h>

#define SIZE	1024

__global__ void vectorAdd(int *a, int *b, int *c, int n) {

	int i = threadIdx.x;

	if (i < n) {
		c[i] = a[i] + b[i];
	}
}

int main() {
	int *a, *b, *c;

	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	vectorAdd <<< 1, SIZE >>>(a, b, c, SIZE);

	hipDeviceSynchronize();

	for (int i = 0; i < 10; i++) {
		printf("c[%d] = %d\n", i, c[i]);
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}